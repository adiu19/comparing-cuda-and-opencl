#include <iostream>
#include <time.h>
#include <random>
#include<hip/hip_runtime.h>
#include <hiprand.h>
#include <math.h>
#include "kernels.cuh"

using namespace std;
int main(){
	unsigned int n = 256*256;
	unsigned int m = 20000;
	int *h_count;
	int *d_count;
	hiprandState *d_state;
	float pi;


	// allocate memory
	h_count = (int*)malloc(n*sizeof(int));
	hipMalloc((void**)&d_count, n*sizeof(int));
	hipMalloc((void**)&d_state, n*sizeof(hiprandState));
	hipMemset(d_count, 0, sizeof(int));


	// set up timing stuff
	float gpu_elapsed_time;
	hipEvent_t gpu_start, gpu_stop;
	hipEventCreate(&gpu_start);
	hipEventCreate(&gpu_stop);
	hipEventRecord(gpu_start, 0);

	dim3 gridSize = 256;
	dim3 blockSize = 256;
	setup_kernel<<< gridSize, blockSize>>>(d_state);

	monti_carlo_pi_kernel<<<gridSize, blockSize>>>(d_state, d_count, m);

	hipMemcpy(h_count, d_count, sizeof(int), hipMemcpyDeviceToHost);
	hipEventRecord(gpu_stop, 0);
	hipEventSynchronize(gpu_stop);
	hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);
	hipEventDestroy(gpu_start);
	hipEventDestroy(gpu_stop);

	pi = *h_count*4.0/(n*m);
	cout<<"Approximate pi calculated on GPU is: "<<pi<<" and calculation took "<<gpu_elapsed_time<<std::endl;

	free(h_count);
	hipFree(d_count);
	hipFree(d_state);
}

