// Approximation of Pi using a simple, and not optimized, CUDA program
#include <iostream>
#include <limits>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

using std::cout;
using std::endl;

typedef unsigned long long Count;
typedef std::numeric_limits<double> DblLim;

const Count TPB = 32;
const Count NBLOCKS = 640;
const Count ITERATIONS = 1000000;

__global__ void picount(Count *totals) {

	__shared__ Count counter[TPB];

	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	hiprandState_t rng;
	hiprand_init(clock64(), tid, 0, &rng);

	counter[threadIdx.x] = 0;

	for (int i = 0; i < ITERATIONS; i++) {
		float x = hiprand_uniform(&rng);
		float y = hiprand_uniform(&rng);
		counter[threadIdx.x] += 1 - int(x * x + y * y);
	}

	if (threadIdx.x == 0) {
		totals[blockIdx.x] = 0;
		for (int i = 0; i < TPB; i++) {
			totals[blockIdx.x] += counter[i];
		}
	}
}

int main(int argc, char **argv) {
	Count *hOut, *dOut;
	hOut = new Count[NBLOCKS];
	hipMalloc(&dOut, sizeof(Count) * NBLOCKS);

	picount<<<NBLOCKS, TPB>>>(dOut);

	hipMemcpy(hOut, dOut, sizeof(Count) * NBLOCKS, hipMemcpyDeviceToHost);
	hipFree(dOut);

	Count total = 0;
	for (int i = 0; i < NBLOCKS; i++) {
		total += hOut[i];
	}
	Count tests = NBLOCKS * ITERATIONS * TPB;
	cout << "Approximated PI using " << tests << " random tests\n";

	cout.precision(DblLim::max_digits10);
	cout << "PI ~= " << 4.0 * (double)total/(double)tests << endl;

	return 0;
}