// Approximation of Pi using a simple, and not optimized, CUDA program
#include <iostream>
#include <limits>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

using namespace std;

typedef unsigned long long Count;
typedef numeric_limits<double> DblLim;

const Count TPB = 32;
const Count NBLOCKS = 65536;
const Count m = 1000000;

__global__ void picount(Count *totals) {

	__shared__ Count counter[TPB];

	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	hiprandState_t rng;
	hiprand_init(clock64(), tid, 0, &rng);

	counter[threadIdx.x] = 0;

	for (int i = 0; i < m; i++) {
		float x = hiprand_uniform(&rng);
		float y = hiprand_uniform(&rng);
		counter[threadIdx.x] += 1 - int(x * x + y * y);
	}

    int i = blockDim.x/2;
	while(i != 0){
		if(threadIdx.x < i){
			counter[threadIdx.x] += counter[threadIdx.x + i];
		}

		i /= 2;
		__syncthreads();
	}

	if (threadIdx.x == 0) {
        atomicAdd(totals, counter[0]);
	}
}

int main(int argc, char **argv) {
	Count *hOut, *dOut;
	hOut = new Count[1];
	hipMalloc(&dOut, sizeof(Count) * 1);

	picount<<<NBLOCKS, TPB>>>(dOut);

	hipMemcpy(hOut, dOut, sizeof(Count) * 1, hipMemcpyDeviceToHost);
	hipFree(dOut);

	Count total = hOut[0];

	Count tests = NBLOCKS * m * TPB;
	cout << "Approximated PI using " << tests << " random tests\n";

	cout.precision(DblLim::max_digits10);
	cout << "PI ~= " << 4.0 * (double)total/(double)tests << endl;

	return 0;
}